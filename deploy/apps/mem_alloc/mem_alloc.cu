#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include <chrono>
#include <thread>

// 模仿 https://github.com/wilicc/gpu-burn 的使用方法，但是不使用GPU计算，只占用显存
size_t parseMemoryArgument(const char* arg, size_t freeMem) {
    char* end;
    double memArg = strtod(arg, &end);

    if (*end == '%') {
        return static_cast<size_t>(freeMem * (memArg / 100.0));
    } else {
        return static_cast<size_t>(memArg * 1024 * 1024);
    }
}

int main(int argc, char **argv) {
    size_t totalMem, freeMem;
    hipMemGetInfo(&freeMem, &totalMem);

    size_t memSize = 0;
    int duration = 0;

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-m") == 0 && i + 1 < argc) {
            memSize = parseMemoryArgument(argv[++i], freeMem);
        } else if (strcmp(argv[i], "-t") == 0 && i + 1 < argc) {
            duration = atoi(argv[++i]);
        }
    }

    if (memSize <= 0 || memSize > freeMem) {
        std::cerr << "Invalid memory size." << std::endl;
        return 1;
    }

    char *memPtr;
    hipMalloc((void**)&memPtr, memSize);
    std::cout << "Allocated " << memSize / (1024 * 1024) << " MB of GPU memory." << std::endl;

    std::cout << "Running for " << duration << " seconds..." << std::endl;
    std::this_thread::sleep_for(std::chrono::seconds(duration));

    hipFree(memPtr);
    return 0;
}